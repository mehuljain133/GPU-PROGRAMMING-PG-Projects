// Unit-I Introduction: Introduction to heterogeneous computing, overview of CUDA C/Python, and kernel-based parallel programming


#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function to add elements of two arrays
__global__ void vectorAdd(const float *A, const float *B, float *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000; // Number of elements in vectors
    size_t size = N * sizeof(float);

    // Allocate host memory (CPU)
    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize input vectors on host
    for (int i = 0; i < N; i++) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory (GPU)
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy input vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel on the GPU: parallel addition
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify the result
    for (int i = 0; i < 10; i++) {
        printf("h_C[%d] = %f\n", i, h_C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
